#include "hip/hip_runtime.h"
#include "CUDADenseGraphAnnealer.h"
#include "DeviceKernels.h"
#include "cub_iterator.cuh"
#include <hipcub/hipcub.hpp>
#include "DeviceSegmentedSum.cuh"


namespace sq = sqaod;
using namespace sqaod_cuda;

template<class real>
CUDADenseGraphAnnealer<real>::CUDADenseGraphAnnealer() {
    m_ = -1;
    annState_ = sq::annNone;
}

template<class real>
CUDADenseGraphAnnealer<real>::CUDADenseGraphAnnealer(Device &device) {
    m_ = -1;
    annState_ = sq::annNone;
    assignDevice(device);
}

template<class real>
CUDADenseGraphAnnealer<real>::~CUDADenseGraphAnnealer() {
    if (dotJq_ != NULL) {
        delete dotJq_;
        dotJq_ = NULL;
    }
}


template<class real>
void CUDADenseGraphAnnealer<real>::assignDevice(Device &device) {
    devStream_ = device.defaultStream();
    devAlloc_ = device.objectAllocator();
    dgFuncs_.assignDevice(device);
    devMath_.assignDevice(device);
    devCopy_.assignDevice(device);
    d_random_.assignDevice(device);
    flipPosBuffer_.assignDevice(device);
    realNumBuffer_.assignDevice(device);

    /* initialize sumJq */
    typedef DeviceSegmentedSumTypeImpl<real, InDotPtr<real>, real*, Offset2way> DotJq;
    dotJq_ = new DotJq(device);
}



template<class real>
void CUDADenseGraphAnnealer<real>::seed(unsigned long seed) {
    d_random_.seed(seed);
    annState_ |= sq::annRandSeedGiven;
}

template<class real>
void CUDADenseGraphAnnealer<real>::getProblemSize(int *N, int *m) const {
    *N = N_;
    *m = m_;
}

template<class real>
void CUDADenseGraphAnnealer<real>::setProblem(const Matrix &W, sq::OptimizeMethod om) {
    throwErrorIf(!isSymmetric(W), "W is not symmetric.");
    N_ = W.rows;
    om_ = om;

    DeviceMatrix *dW = devStream_->tempDeviceMatrix<real>(W.dim(), __func__);
    devCopy_(dW, W);
    if (om == sq::optMaximize)
        devMath_.scale(dW, -1., *dW);
    dgFuncs_.calculate_hJc(&d_h_, &d_J_, &d_c_, *dW);
}

template<class real>
void CUDADenseGraphAnnealer<real>::setNumTrotters(int m) {
    throwErrorIf(m <= 0, "# trotters must be a positive integer.");
    m_ = m;
    HostObjectAllocator halloc;
    devAlloc_->allocate(&d_matq_, m_, N_);
    devAlloc_->allocate(&d_Jq_, m_);
    halloc.allocate(&h_E_, m_);
    halloc.allocate(&h_q_, sq::Dim(m_, N_));
    xlist_.reserve(m);
    qlist_.reserve(m);
    /* estimate # rand nums required per one anneal. */
    int requiredSize = (N_ * m_ * (nRunsPerRandGen + 1)) * sizeof(real) / 4;
    d_random_.setRequiredSize(requiredSize);

    typedef DeviceSegmentedSumTypeImpl<real, InDotPtr<real>, real*, Offset2way> DotJq;
    DotJq &dotJq = static_cast<DotJq&>(*dotJq_);
    dotJq.configure(N_, m_, false);

    annState_ |= annNTrottersGiven;
}

template<class real>
void CUDADenseGraphAnnealer<real>::get_hJc(Vector *h, Matrix *J, real *c) const {
    devCopy_(h, d_h_);
    devCopy_(J, d_J_);
    devCopy_(c, d_c_);
    devCopy_.synchronize();
}

template<class real>
void CUDADenseGraphAnnealer<real>::randomize_q() {
    /* FIXME: add exception, randomize_q() must be called after calling seed() and setNumTrotters(). */
    ::randomize_q(d_matq_.d_data, d_random_, d_matq_.rows * d_matq_.cols,
                  devStream_->getCudaStream());
}

template<class real>
void CUDADenseGraphAnnealer<real>::calculate_E() {
    dgFuncs_.calculate_E(&h_E_, d_h_, d_J_, d_c_, d_matq_);
}

template<class real>
void CUDADenseGraphAnnealer<real>::initAnneal() {
    if (!(annState_ & annNTrottersGiven))
        setNumTrotters((N_) / 4);
    annState_ |= annNTrottersGiven;
    if (!(annState_ & annRandSeedGiven))
        d_random_.seed();
    annState_ |= annRandSeedGiven;
    if (!(annState_ & annQSet))
        randomize_q();
    annState_ |= annQSet;
}

template<class real>
void CUDADenseGraphAnnealer<real>::finAnneal() {
    devStream_->synchronize();
    syncBits();
    calculate_E();
    devStream_->synchronize();
    E_.map(h_E_.d_data, h_E_.size);
}

template<class real>
void CUDADenseGraphAnnealer<real>::syncBits() {
    xlist_.clear();
    qlist_.clear();

    for (int idx = 0; idx < sq::IdxType(m_); ++idx) {
        Bits q(h_q_.row(idx), N_);
        qlist_.pushBack(q);
        Bits x(sqaod::SizeType(qlist_.size()));
        x = x_from_q(q);
        xlist_.pushBack(x);
    }
}

#if 0
/* equivalent code */
template<class real>
void annealOneStep(real G, real kT) {
    real twoDivM = real(2.) / real(m_);
    real coef = std::log(std::tanh(G / kT / m_)) / kT;

    for (int outer = 0; outer < IdxType(N_); ++outer) {
        int x[m];

        /* carried out in DeviceRandomBuffer. */
        for (int y = 0; y < IdxType(m_); ++y) {
            /* first plane */
            int fraction = y % 2;
            /* second plane */
            int fraction = 1 - y % 2;

            x[innder] = (random_random() * 2 + fraction) % N;
        }

        /* calculate_Jq() */
        real d_Jq[m];
        for (int y = 0; y < IdxType(m_); ++y)
            d_Jq[y] = J_.row(x[y]).dot(matQ_.row(y));

        /* flip each bit, */
        for (int inner = 0; inner < IdxType(m_); ++inner) {
            /* flip one bit */
            real qyx = matQ_(y, x[m]);

            real dE = - twoDivM * qyx * (d_Jq[x[y] + h_(x[y])];
            int neibour0 = (m_ + y - 1) % m_, neibour1 = (y + 1) % m_;
            dE -= qyx * (matQ_(neibour0, x) + matQ_(neibour1, x)) * coef;
            real threshold = (dE < real(0.)) ? real(1.) : std::exp(-dE / kT);
            if (threshold > random_.random<real>())
                matQ_(y, x) = - qyx;
        }
    }
}
#endif

template<class real>
void CUDADenseGraphAnnealer<real>::calculate_Jq(DeviceVector *d_Jq,
                                                const DeviceMatrix &d_J, const DeviceMatrix &d_matq,
                                                const int *d_flipPos) {
    hipStream_t stream = devStream_->getCudaStream();
    InDotPtr<real> inPtr(d_matq.d_data, d_J.d_data);
    typedef DeviceSegmentedSumTypeImpl<real, InDotPtr<real>, real*, Offset2way> DotJq;
    DotJq &dotJq = static_cast<DotJq&>(*dotJq_);
    dotJq(inPtr, d_Jq->d_data, Offset2way(d_flipPos, N_));
}

template<class real>
__global__ static void
tryFlipKernel(real *d_q, const real *d_Jq, const real *d_h,
              const int *d_x, const real *d_random, sq::SizeType N, sq::SizeType m,
             const real twoDivM, const real coef, const real invKT) {
    int y = blockDim.x * blockIdx.x + threadIdx.x; /* m */
    if (y < m) {
        int x = d_x[y]; /* N */
        real qyx = d_q[N * y + x];

        int neibour0 = (y == 0) ? m - 1 : y - 1;
        int neibour1 = (y == m - 1) ? 0 : y + 1;

        real dE = - twoDivM * qyx * (d_Jq[y] + d_h[x]);
        dE -= qyx * (d_q[N * neibour0 + x] + d_q[N * neibour1 + x]) * coef;
        real threshold = (dE < real(0.)) ? real(1.) : exp(- dE * invKT);
        if (threshold > d_random[y])
            d_q[N * y + x] = - qyx;
    }
}

template<class real> void CUDADenseGraphAnnealer<real>::
annealOneStep(DeviceMatrix *d_matq, const DeviceVector &d_Jq, const int *d_x, const real *d_random,
              const DeviceVector &d_h, const DeviceMatrix &d_J, real G, real kT) {
    real twoDivM = real(2.) / real(m_);
    real coef = std::log(std::tanh(G / kT / m_)) / kT;
    real invKT = real(1.) / kT;

    dim3 blockDim(128);
    dim3 gridDim(divru((sq::SizeType)m_, blockDim.x));
    tryFlipKernel<<<gridDim, blockDim>>>(d_matq->d_data, d_Jq.d_data, d_h.d_data,
                                         d_x, d_random, N_, m_,
                                         twoDivM, coef, invKT);
    DEBUG_SYNC;
}



template<class real>
void CUDADenseGraphAnnealer<real>::annealOneStep(real G, real kT) {
    if (!flipPosBuffer_.available(m_ * N_))
        flipPosBuffer_.generateFlipPositions(d_random_, N_, m_, nRunsPerRandGen);
    if (!realNumBuffer_.available(m_ * N_))
        realNumBuffer_.generate<real>(d_random_, N_ * m_ * nRunsPerRandGen);
    for (int idx = 0; idx < N_; ++idx) {
        const int *d_flipPos = flipPosBuffer_.acquire<int>(m_);
        const real *d_random = realNumBuffer_.acquire<real>(m_);
        calculate_Jq(&d_Jq_, d_J_, d_matq_, d_flipPos);
        annealOneStep(&d_matq_, d_Jq_, d_flipPos, d_random, d_h_, d_J_, G, kT);
    }
}




template class CUDADenseGraphAnnealer<double>;
template class CUDADenseGraphAnnealer<float>;
