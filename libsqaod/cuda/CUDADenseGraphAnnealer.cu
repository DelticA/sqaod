#include "hip/hip_runtime.h"
#include "CUDADenseGraphAnnealer.h"
#include "DeviceKernels.h"
#include "cub_iterator.cuh"
#include <hipcub/hipcub.hpp>
#include "DeviceSegmentedSum.cuh"


namespace sq = sqaod;
using namespace sqaod_cuda;

template<class real>
CUDADenseGraphAnnealer<real>::CUDADenseGraphAnnealer() {
    devStream_ = NULL;
    m_ = (SizeType)-1;
}

template<class real>
CUDADenseGraphAnnealer<real>::CUDADenseGraphAnnealer(Device &device) {
    devStream_ = NULL;
    m_ = (SizeType)-1;
    assignDevice(device);
}

template<class real>
CUDADenseGraphAnnealer<real>::~CUDADenseGraphAnnealer() {
    if (annState_ & annInitialized)
        deallocate();
    d_random_.deallocate();
    if (dotJq_ != NULL) {
        delete dotJq_;
        dotJq_ = NULL;
    }
}

template<class real>
void CUDADenseGraphAnnealer<real>::deallocate() {
    if (annState_ & annInitialized) {
        devAlloc_->deallocate(d_J_);
        devAlloc_->deallocate(d_h_);
        devAlloc_->deallocate(d_c_);
        devAlloc_->deallocate(d_matq_);
        devAlloc_->deallocate(d_Jq_);
        
        HostObjectAllocator halloc;
        halloc.deallocate(h_E_);
        halloc.deallocate(h_q_);
        E_ = HostVector();
        
        flipPosBuffer_.deallocate();
        realNumBuffer_.deallocate();

        annState_ &= ~(int)(annInitialized | annQSet);
    }
}

template<class real>
void CUDADenseGraphAnnealer<real>::assignDevice(Device &device) {
    throwErrorIf(devAlloc_ != NULL, "Device assigned more than once.");
    devStream_ = device.defaultStream();
    devAlloc_ = device.objectAllocator();
    devFormulas_.assignDevice(device);
    devCopy_.assignDevice(device);
    d_random_.assignDevice(device);
    flipPosBuffer_.assignDevice(device);
    realNumBuffer_.assignDevice(device);

    /* initialize sumJq */
    typedef DeviceSegmentedSumTypeImpl<real, InDotPtr<real>, real*, Offset2way> DotJq;
    dotJq_ = new DotJq(device);
}

template<class real>
sq::Algorithm CUDADenseGraphAnnealer<real>::selectAlgorithm(Algorithm algo) {
    return sq::algoColoring;
}

template<class real>
sq::Algorithm CUDADenseGraphAnnealer<real>::getAlgorithm() const {
    return sq::algoColoring;
}


template<class real>
void CUDADenseGraphAnnealer<real>::seed(unsigned int seed) {
    throwErrorIf(devStream_ == NULL, "Device not set.");
    d_random_.seed(seed);
    annState_ |= sq::annRandSeedGiven;
}

template<class real>
void CUDADenseGraphAnnealer<real>::setProblem(const HostMatrix &W, sq::OptimizeMethod om) {
    throwErrorIf(!isSymmetric(W), "W is not symmetric.");
    if (W.rows != N_)
        deallocate();

    N_ = W.rows;
    m_ = N_ / 4;
    om_ = om;

    DeviceMatrix *dW = devStream_->tempDeviceMatrix<real>(W.dim(), __func__);
    devCopy_(dW, W);
    if (om == sq::optMaximize)
        devFormulas_.devMath.scale(dW, -1., *dW);
    devFormulas_.calculate_hJc(&d_h_, &d_J_, &d_c_, *dW);

    annState_ |= annProblemSet;
}


template<class real>
void CUDADenseGraphAnnealer<real>::set_x(const Bits &x) {
    throwErrorIf((annState_ & annQSetReady) == annQSetReady,
                 "set_x() must be called after initAnneal()");
    /* FIXME: add size check */
    HostVector rx = sq::x_to_q<real>(x);
    DeviceVector *d_x = devStream_->tempDeviceVector<real>(rx.size);
    devCopy_(d_x, rx);
    devFormulas_.devMath.scaleBroadcast(&d_matq_, real(1.), *d_x, opRowwise);
    annState_ |= annQSet;
}


template<class real>
void CUDADenseGraphAnnealer<real>::get_hJc(HostVector *h, HostMatrix *J, real *c) const {
    throwErrorIf(!(annState_ & annProblemSet), "Problem unset.");

    devCopy_(h, d_h_);
    devCopy_(J, d_J_);
    devCopy_(c, d_c_);
    devCopy_.synchronize();
}

template<class real>
void CUDADenseGraphAnnealer<real>::randomize_q() {
    throwErrorIf((annState_ & annQSetReady) == annQSetReady,
                 "randomize_q() must be called after initAnneal()");

    ::randomize_q(d_matq_.d_data, d_random_, d_matq_.rows * d_matq_.cols,
                  devStream_->getCudaStream());
   annState_ |= annQSet;
}

template<class real>
void CUDADenseGraphAnnealer<real>::calculate_E() {
    throwErrorIf((annState_ & annQSet) == 0, "q is not initialized.");

    DeviceVector *d_E = devStream_->tempDeviceVector<real>(m_);
    devFormulas_.calculate_E(d_E, d_h_, d_J_, d_c_, d_matq_);
    real sign = (om_ == optMaximize) ? -1. : 1.;
    devFormulas_.devMath.scale(&h_E_, sign, *d_E);
}

template<class real>
void CUDADenseGraphAnnealer<real>::initAnneal() {
    if (!(annState_ & annRandSeedGiven))
        d_random_.seed();
    annState_ |= annRandSeedGiven;

    if (annState_ & annInitialized)
        deallocate();
    
    HostObjectAllocator halloc;
    devAlloc_->allocate(&d_matq_, m_, N_);
    devAlloc_->allocate(&d_Jq_, m_);
    halloc.allocate(&h_E_, m_);
    halloc.allocate(&h_q_, sq::Dim(m_, N_));
    xlist_.reserve(m_);
    qlist_.reserve(m_);
    /* estimate # rand nums required per one anneal. */
    int requiredSize = (N_ * m_ * (nRunsPerRandGen + 1)) * sizeof(real) / 4;
    d_random_.setRequiredSize(requiredSize);
    
    typedef DeviceSegmentedSumTypeImpl<real, InDotPtr<real>, real*, Offset2way> DotJq;
    DotJq &dotJq = static_cast<DotJq&>(*dotJq_);
    dotJq.configure(N_, m_, false);
}

template<class real>
void CUDADenseGraphAnnealer<real>::finAnneal() {
    throwErrorIf((annState_ & annInitialized) == 0, "not initialized.");

    devStream_->synchronize();
    syncBits();
    calculate_E();
    devStream_->synchronize();
    E_.map(h_E_.d_data, h_E_.size);
}

template<class real>
void CUDADenseGraphAnnealer<real>::syncBits() {
    xlist_.clear();
    qlist_.clear();

    devFormulas_.devMath.toBits(&h_q_, d_matq_);
    devStream_->synchronize();
    for (int idx = 0; idx < sq::IdxType(m_); ++idx) {
        Bits q(h_q_.row(idx), N_);
        qlist_.pushBack(q);
        Bits x(sqaod::SizeType(qlist_.size()));
        x = x_from_q(q);
        xlist_.pushBack(x);
    }
}

#if 0
/* equivalent code */
template<class real>
void annealOneStep(real G, real kT) {
    real twoDivM = real(2.) / real(m_);
    real coef = std::log(std::tanh(G / kT / m_)) / kT;

    for (int outer = 0; outer < IdxType(N_); ++outer) {
        int x[m];

        /* carried out in DeviceRandomBuffer. */
        for (int y = 0; y < IdxType(m_); ++y) {
            /* first plane */
            int fraction = y % 2;
            /* second plane */
            int fraction = 1 - y % 2;

            x[innder] = (random_random() * 2 + fraction) % N;
        }

        /* calculate_Jq() */
        real d_Jq[m];
        for (int y = 0; y < IdxType(m_); ++y)
            d_Jq[y] = J_.row(x[y]).dot(matQ_.row(y));

        /* flip each bit, */
        for (int inner = 0; inner < IdxType(m_); ++inner) {
            /* flip one bit */
            real qyx = matQ_(y, x[m]);

            real dE = - twoDivM * qyx * (d_Jq[x[y] + h_(x[y])];
            int neibour0 = (m_ + y - 1) % m_, neibour1 = (y + 1) % m_;
            dE -= qyx * (matQ_(neibour0, x) + matQ_(neibour1, x)) * coef;
            real threshold = (dE < real(0.)) ? real(1.) : std::exp(-dE / kT);
            if (threshold > random_.random<real>())
                matQ_(y, x) = - qyx;
        }
    }
}
#endif

template<class real>
void CUDADenseGraphAnnealer<real>::calculate_Jq(DeviceVector *d_Jq,
                                                const DeviceMatrix &d_J, const DeviceMatrix &d_matq,
                                                const int *d_flipPos) {
    hipStream_t stream = devStream_->getCudaStream();
    InDotPtr<real> inPtr(d_matq.d_data, d_J.d_data);
    typedef DeviceSegmentedSumTypeImpl<real, InDotPtr<real>, real*, Offset2way> DotJq;
    DotJq &dotJq = static_cast<DotJq&>(*dotJq_);
    dotJq(inPtr, d_Jq->d_data, Offset2way(d_flipPos, N_));
}

template<class real>
__global__ static void
tryFlipKernel(real *d_q, const real *d_Jq, const real *d_h,
              const int *d_x, const real *d_random, sq::SizeType N, sq::SizeType m,
             const real twoDivM, const real coef, const real invKT) {
    int y = blockDim.x * blockIdx.x + threadIdx.x; /* m */
    if (y < m) {
        int x = d_x[y]; /* N */
        real qyx = d_q[N * y + x];

        int neibour0 = (y == 0) ? m - 1 : y - 1;
        int neibour1 = (y == m - 1) ? 0 : y + 1;

        real dE = - twoDivM * qyx * (d_Jq[y] + d_h[x]);
        dE -= qyx * (d_q[N * neibour0 + x] + d_q[N * neibour1 + x]) * coef;
        real threshold = (dE < real(0.)) ? real(1.) : exp(- dE * invKT);
        if (threshold > d_random[y])
            d_q[N * y + x] = - qyx;
    }
}

template<class real> void CUDADenseGraphAnnealer<real>::
annealOneStep(DeviceMatrix *d_matq, const DeviceVector &d_Jq, const int *d_x, const real *d_random,
              const DeviceVector &d_h, const DeviceMatrix &d_J, real G, real kT) {
    real twoDivM = real(2.) / real(m_);
    real coef = std::log(std::tanh(G / kT / m_)) / kT;
    real invKT = real(1.) / kT;

    dim3 blockDim(128);
    dim3 gridDim(divru((sq::SizeType)m_, blockDim.x));
    tryFlipKernel<<<gridDim, blockDim>>>(d_matq->d_data, d_Jq.d_data, d_h.d_data,
                                         d_x, d_random, N_, m_,
                                         twoDivM, coef, invKT);
    DEBUG_SYNC;
}



template<class real>
void CUDADenseGraphAnnealer<real>::annealOneStep(real G, real kT) {
    throwErrorIf((annState_ & annQSet) == 0, "q is not initialized.");

    if (!flipPosBuffer_.available(m_ * N_))
        flipPosBuffer_.generateFlipPositions(d_random_, N_, m_, nRunsPerRandGen);
    if (!realNumBuffer_.available(m_ * N_))
        realNumBuffer_.generate<real>(d_random_, N_ * m_ * nRunsPerRandGen);
    for (int idx = 0; idx < N_; ++idx) {
        const int *d_flipPos = flipPosBuffer_.acquire<int>(m_);
        const real *d_random = realNumBuffer_.acquire<real>(m_);
        calculate_Jq(&d_Jq_, d_J_, d_matq_, d_flipPos);
        annealOneStep(&d_matq_, d_Jq_, d_flipPos, d_random, d_h_, d_J_, G, kT);
    }
}




template class CUDADenseGraphAnnealer<double>;
template class CUDADenseGraphAnnealer<float>;
