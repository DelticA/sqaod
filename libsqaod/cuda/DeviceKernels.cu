#include "hip/hip_runtime.h"
#include "cudafuncs.h"
#include "DeviceKernels.h"
#include <hipcub/hipcub.hpp>

using sqaod::SizeType;
using sqaod::IdxType;
using namespace sqaod_cuda;

#ifdef _DEBUG
#define CUB_DEBUG (true)
#else
#define CUB_DEBUG (false)
#endif


/* FIXME: add __forceinline__ for device funcs/methods. */

namespace {
        
template<class real>
struct AddAssign {
    __device__ AddAssign(real &_d_value, real _mulFactor, real _alpha) : d_value(_d_value), mulFactor(_mulFactor), alpha(_alpha) { }
    __forceinline__
    __device__ real operator=(const real &v) const {
        return d_value = mulFactor * d_value + alpha * v;
    }
    real &d_value;
    real mulFactor;
    real alpha;
};

template<class real>
struct AddAssignDevPtr {
    typedef real value_type;

    AddAssignDevPtr(real *_d_data, real _mulFactor, real _alpha) : d_data(_d_data), mulFactor(_mulFactor), alpha(_alpha) { }
    typedef AddAssign<real> Ref;
    __device__ Ref operator*() const {
        return Ref(*d_data, mulFactor, alpha);
    }
    __device__ Ref operator[](SizeType idx) const {
        return Ref(d_data[idx], mulFactor, alpha);
    }

    real *d_data;
    real mulFactor;
    real alpha;
};


template<class real>
struct StridedInPtr {
    typedef real value_type;
    typedef StridedInPtr SelfType;
    __host__ __device__
    StridedInPtr(const real *_d_data, SizeType _stride, IdxType _offset) : d_data(_d_data), stride(_stride), offset(_offset) { }
    typedef AddAssign<real> Ref;
    __device__ const real &operator[](SizeType idx) const {
        return d_data[offset + idx * stride];
    }
    __device__
    SelfType operator+(IdxType v) const {
        return SelfType(d_data + v, stride, offset);
    }

    const real *d_data;
    SizeType stride;
    IdxType offset;
};


template<class type, class real>
struct dev_iterator_traits {
    using difference_type   = ptrdiff_t;
    typedef real              value_type;
    using pointer           = real*;
    using reference         = real&;
    using iterator_category = std::random_access_iterator_tag;
};


}

namespace std {

template<class real>
struct iterator_traits<AddAssignDevPtr<real> > : dev_iterator_traits<AddAssignDevPtr<real>, real> { };
template<class real>
struct iterator_traits<StridedInPtr<real>> : dev_iterator_traits<StridedInPtr<real>, real> { };

}




template<class OutType, class real>  static __global__
void scaleKernel(OutType d_y, real alpha, const real *d_x, SizeType size) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < size)
        d_y[gid] = alpha * d_x[gid];
}

template<class real>
void DeviceMathKernelsType<real>::scale(real *d_y, real alpha, const real *d_x, SizeType size, real addAssignFactor) {
    dim3 blockDim(128);
    dim3 gridDim(divru(size, blockDim.x));
    if (addAssignFactor == 0.) {
        scaleKernel <<<gridDim, blockDim, 0, stream_ >>> (d_y, alpha, d_x, size);
    }
    else {
        AddAssignDevPtr<real> outPtr(d_y, addAssignFactor, real(1.));
        scaleKernel <<<gridDim, blockDim, 0, stream_ >>> (outPtr, alpha, d_x, size);
    }
    DEBUG_SYNC;
}

template<class real, class OutType>
static __global__
void scaleBroadcastKernel(OutType d_y, real alpha, const real *d_c, SizeType size) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < size)
        d_y[gid] = alpha * (*d_c);
}

template<class real> void DeviceMathKernelsType<real>::
scaleBroadcast(real *d_y, real alpha, const real *d_c, SizeType size,
               real addAssignFactor) {
    dim3 blockDim(128);
    dim3 gridDim(divru(size, blockDim.x));
    if (addAssignFactor == 0.) {
        scaleBroadcastKernel<<<gridDim, blockDim, 0, stream_>>>
                (d_y, alpha, d_c, size);
    }
    else {
        AddAssignDevPtr<real> outPtr(d_y, addAssignFactor, real(1.));
        scaleBroadcastKernel<real><<<gridDim, blockDim, 0, stream_>>>(outPtr, alpha, d_c, size);
    }
    DEBUG_SYNC;
}

template<class real, class OutPtrType>  static __global__
void scaleBroadcastVectorKernel(OutPtrType d_A, real alpha, const real *d_x, SizeType size) {
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int gidy = blockDim.y * blockIdx.y + threadIdx.y;
    if (gidx < size) {
        SizeType pos = gidx + size * gidy;
        d_A[pos] = alpha * d_x[gidx];
    }
}

template<class real>
void DeviceMathKernelsType<real>::
scaleBroadcastVector(real *d_A, real alpha, const real *d_x, SizeType size,
                     SizeType nBatch, real addAssignFactor) {
    dim3 blockDim(128);
    dim3 gridDim(divru(size, blockDim.x), divru(nBatch, blockDim.y));
    if (addAssignFactor == 0.) {
        scaleBroadcastVectorKernel<<<gridDim, blockDim, 0, stream_>>>(d_A, alpha, d_x, size);
    }
    else {
        AddAssignDevPtr<real> outPtr(d_A, addAssignFactor, real(1.));
        scaleBroadcastVectorKernel<<<gridDim, blockDim, 0, stream_>>>(outPtr, alpha, d_x, size);
    }
    DEBUG_SYNC;
}


template<class real, class OutPtrType>
static __global__
void scaleBroadcastScalarsKernel(OutPtrType d_A, real alpha, const real *d_x, SizeType size) {
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int gidy = blockDim.y * blockIdx.y + threadIdx.y;
    if (gidx < size) {
        SizeType pos = gidx + size * gidy;
        d_A[pos] = alpha * d_x[gidy];
    }
}

template<class real>
void DeviceMathKernelsType<real>::
scaleBroadcastScalars(real *d_A, real alpha, const real *d_x, SizeType size,
                     SizeType nBatch, real addAssignFactor) {
    dim3 blockDim(128);
    dim3 gridDim(divru(size, blockDim.x), divru(nBatch, blockDim.y));
    if (addAssignFactor == 0.) {
        scaleBroadcastScalarsKernel
                <<<gridDim, blockDim, 0, stream_>>>(d_A, alpha, d_x, size);
    }
    else {
        AddAssignDevPtr<real> outPtr(d_A, addAssignFactor, real(1.));
        scaleBroadcastScalarsKernel
                <<<gridDim, blockDim, 0, stream_>>>(outPtr, alpha, d_x, size);
    }
    DEBUG_SYNC;
}


template<class real> void DeviceMathKernelsType<real>::
sum(real *d_sum, real alpha, const real *d_x, SizeType size, real addAssignFactor) {
    size_t temp_storage_bytes;
    hipcub::DeviceReduce::Sum(NULL, temp_storage_bytes,
                           d_x, d_sum, size, stream_, CUB_DEBUG);
    void *d_temp_storage = devStream_->allocate(temp_storage_bytes, __func__);
    if (addAssignFactor == 0.) {
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               d_x, d_sum, size, stream_, CUB_DEBUG);
    }
    else {
        AddAssignDevPtr<real> outPtr(d_sum, addAssignFactor, real(1.));
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               d_x, outPtr, size, stream_, CUB_DEBUG);
    }
}



template<class real> void DeviceMathKernelsType<real>::
sumGather(real *d_sum, real alpha, const real *d_x, SizeType size, SizeType stride, int offset) {
    size_t temp_storage_bytes;
    hipcub::DeviceReduce::Sum(NULL, temp_storage_bytes,
                           d_x, d_sum, size, stream_, CUB_DEBUG);
    void *d_temp_storage = devStream_->allocate(temp_storage_bytes, __func__);
    StridedInPtr<real> inPtr(d_x, stride, offset);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                           inPtr, d_sum, size, stream_, CUB_DEBUG);
}


namespace {
/* Functors for offsets */

struct Linear {
    Linear(IdxType _a, IdxType _b) : a(_a), b(_b) { }
    __device__
    IdxType operator[](IdxType idx) const { return a * idx + b; }
    IdxType a, b;
};

}

template<class real> void DeviceMathKernelsType<real>::
sumBatched(real *d_sum, real alpha, const real *d_A, SizeType size, SizeType nBatch) {
    size_t temp_storage_bytes;
    hipcub::DeviceSegmentedReduce::Sum(NULL, temp_storage_bytes,
                                    d_A, d_sum, nBatch,
                                    Linear(size, 0), Linear(size, size),
                                    stream_, CUB_DEBUG);
    void *d_temp_storage = devStream_->allocate(temp_storage_bytes, __func__);
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes,
                                    d_A, d_sum, nBatch,
                                    Linear(size, 0), Linear(size, size),
                                    stream_, CUB_DEBUG);
    DEBUG_SYNC;
}


namespace {

template<class real>
struct InDotPtr {
    typedef InDotPtr<real> SelfType;
    
    __host__ __device__
    InDotPtr(const real *_d_x, const real *_d_y) : d_x(_d_x), d_y(_d_y) { }
    __device__
    real operator[](IdxType idx) const {
        return d_x[idx] * d_y[idx];
    }
    __device__
    SelfType operator+(IdxType idx) const {
        return SelfType(&d_x[idx], &d_y[idx]);
    }
    
    const real *d_x, *d_y;
};

}

namespace std {

template<class real>
struct iterator_traits<InDotPtr<real>> : dev_iterator_traits<InDotPtr<real>, real> { };

}


template<class real> void DeviceMathKernelsType<real>::
dot(real *d_c, real alpha, const real *d_x, const real *d_y, SizeType size,
    real addAssignFactor) {

    InDotPtr<real> inPtr(d_x, d_y);
    size_t temp_storage_bytes;
    hipcub::DeviceReduce::Sum(NULL, temp_storage_bytes,
                           inPtr, d_c, size, stream_, CUB_DEBUG);
    void *d_temp_storage = devStream_->allocate(temp_storage_bytes, __func__);

    if (addAssignFactor == 0.) {
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               inPtr, d_c, size, stream_, CUB_DEBUG);
    }
    else {
        AddAssignDevPtr<real> outPtr(d_c, addAssignFactor, real(1.));
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               inPtr, outPtr, size, stream_, CUB_DEBUG);
    }
    DEBUG_SYNC;
}


template<class real> void DeviceMathKernelsType<real>::
dotBatched(real *d_z, real alpha, const real *d_x, const real *d_y, SizeType size,
           SizeType nBatch) {
    
    InDotPtr<real> inPtr(d_x, d_y);
    
    size_t temp_storage_bytes;
    hipcub::DeviceSegmentedReduce::Sum(NULL, temp_storage_bytes,
                                    inPtr, d_z, nBatch,
                                    Linear(size, 0), Linear(size, size),
                                    stream_, CUB_DEBUG);
    void *d_temp_storage = devStream_->allocate(temp_storage_bytes, __func__);
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes,
                                    inPtr, d_z, nBatch,
                                    Linear(size, 0), Linear(size, size),
                                    stream_, CUB_DEBUG);
    DEBUG_SYNC;
}

template <class real>
__global__ static void
transposeKernel(real *d_At, const real *d_A, SizeType rows, SizeType cols) {


    int inTileLeft = blockDim.x * blockIdx.x * 32;
    int inTileTop = blockDim.y * blockIdx.y * 32;
    
    int xIn = inTileLeft + threadIdx.x;
    int yIn = inTileTop + threadIdx.y;

    real vIn = (xIn < cols) && (yIn < rows) ? d_A[xIn + cols * yIn] : real();

    __shared__ real tile[32][33];
    tile[threadIdx.y][threadIdx.x] = vIn;
	__syncthreads();

    int xOut = inTileTop + threadIdx.x;
    int yOut = inTileLeft + threadIdx.y;
    real vOut = tile[threadIdx.x][threadIdx.y];
    
    if ((xOut < cols) && (yOut < rows))
        d_At[xOut + cols * yOut] = vOut;
}


template<class real> void DeviceMathKernelsType<real>::
transpose(real *d_At, const real *d_A, SizeType rows, SizeType cols) {
    dim3 blockDim(32, 32);
    dim3 gridDim(divru(rows, 32u), divru(cols, 32u));
    transposeKernel<<<gridDim, blockDim, 0, stream_>>>(d_At, d_A, rows, cols);
    DEBUG_SYNC;
}


template<class real> void DeviceMathKernelsType<real>::
min(real *d_min, const real *d_values, SizeType size) {
    size_t temp_storage_bytes;
    hipcub::DeviceReduce::Min(NULL, temp_storage_bytes,
                           d_values, d_min, size, stream_, CUB_DEBUG);
    void *d_temp_storage = devStream_->allocate(temp_storage_bytes, __func__);
    hipcub::DeviceReduce::Min(d_temp_storage, temp_storage_bytes,
                           d_values, d_min, size, stream_, CUB_DEBUG);
    DEBUG_SYNC;
}

template<> void DeviceMathKernelsType<double>::
gemv(hipblasOperation_t op, int M, int N,
     const double *d_alpha, const double *d_A, const double *d_x,
     const double *d_beta, double *d_y) {
    throwOnError(hipblasDgemv(devStream_->getCublasHandle(), op, M, N, d_alpha, d_A, N, d_x, 1, d_beta, d_y, 1));
}

template<> void DeviceMathKernelsType<float>::
gemv(hipblasOperation_t op, int M, int N,
     const float *d_alpha, const float *d_A, const float *d_x,
     const float *d_beta, float *d_y) {
    throwOnError(hipblasSgemv(devStream_->getCublasHandle(), op, M, N, d_alpha, d_A, N, d_x, 1, d_beta, d_y, 1));
}

template<> void DeviceMathKernelsType<double>::
gemm(hipblasOperation_t opA, hipblasOperation_t opB, int M, int N, int K,
     const double *d_alpha, const double *d_A, const double *d_B,
     const double *d_beta, double *d_C) {
    throwOnError(hipblasDgemm(devStream_->getCublasHandle(), opA, opB, M, N, K, d_alpha, d_A, M, d_B, K, d_beta, d_C, M));
}

template<> void DeviceMathKernelsType<float>::
gemm(hipblasOperation_t opA, hipblasOperation_t opB, int M, int N, int K,
     const float *d_alpha, const float *d_A, const float *d_B,
     const float *d_beta, float *d_C) {
    throwOnError(hipblasSgemm(devStream_->getCublasHandle(), opA, opB, M, N, K, d_alpha, d_A, M, d_B, K, d_beta, d_C, M));
}

template<class real> DeviceMathKernelsType<real>::
DeviceMathKernelsType(DeviceStream *devStream) {
    devStream_ = devStream;
    stream_ = NULL;
    if (devStream != NULL)
        setStream(devStream);
}

template<class real> void DeviceMathKernelsType<real>::
setStream(DeviceStream *devStream) {
    devStream_ = devStream;
    stream_ = NULL;
    if (devStream_ != NULL)
        stream_ = devStream_->getCudaStream();
}

template struct sqaod_cuda::DeviceMathKernelsType<double>;
template struct sqaod_cuda::DeviceMathKernelsType<float>;


/* DeviceCopyKernels */

template<class V>
__global__ static
void copyBroadcastKernel(V *d_buf, const V v, SizeType size) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < size)
        d_buf[gid] = v;
}


template<class V> void DeviceCopyKernels::
copyBroadcast(V *d_buf, const V &v, sqaod::SizeType size) const {
    dim3 blockDim(128);
    dim3 gridDim(divru(size, blockDim.x));
    copyBroadcastKernel<<<gridDim, blockDim>>>(d_buf, v, size);
    DEBUG_SYNC;
}


template<class V>
__global__ static
void copyBroadcastStridedKernel(V *d_buf, const V v, SizeType size, SizeType stride, IdxType offset) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < size) {
        IdxType pos = gid * stride + offset;
        d_buf[pos] = v;
    }
}


template<class V> void DeviceCopyKernels::
copyBroadcastStrided(V *d_buf, const V &v, SizeType size, SizeType stride, IdxType offset) const {
    dim3 blockDim(128);
    dim3 gridDim(divru(size, blockDim.x));
    copyBroadcastStridedKernel<<<gridDim, blockDim>>>(d_buf, v, size, stride, offset);
    DEBUG_SYNC;
}


void DeviceCopyKernels::setCUDAStream(hipStream_t stream) {
    stream_ = stream;
}


template void DeviceCopyKernels::copyBroadcastStrided(double *, const double &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(float *, const float &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(char *, const char &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(unsigned char *, const unsigned char &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(short *, const short &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(unsigned short *, const unsigned short &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(int *, const int &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(unsigned int *, const unsigned int &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(long *, const long &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(unsigned long *, const unsigned long &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(long long *, const long long &, SizeType, SizeType, IdxType) const;
template void DeviceCopyKernels::copyBroadcastStrided(unsigned long long *, const unsigned long long &, SizeType, SizeType, IdxType) const;

template void DeviceCopyKernels::copyBroadcast(double *, const double &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(float *, const float &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(char *, const char &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(unsigned char *, const unsigned char &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(short *, const short &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(unsigned short *, const unsigned short &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(int *, const int &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(unsigned int *, const unsigned int &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(long *, const long &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(unsigned long *, const unsigned long &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(long long *, const long long &, SizeType) const;
template void DeviceCopyKernels::copyBroadcast(unsigned long long *, const unsigned long long &, SizeType) const;
