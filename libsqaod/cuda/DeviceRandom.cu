#include "hip/hip_runtime.h"
#include "DeviceRandom.h"
#include "cudafuncs.h"
#include <hiprand/hiprand_mtgp32_host.h>
#include <hiprand/hiprand_kernel.h>


/* FIXME: use multiple states utilize more threads.
 *         use memory store */

using namespace sqaod_cuda;
namespace sq = sqaod;
    
enum {
    randGenSize = CURAND_NUM_MTGP32_PARAMS * THREAD_NUM
};


DeviceRandom::DeviceRandom(Device &device, DeviceStream *devStream) {
    assignDevice(device, devStream);
}

DeviceRandom::DeviceRandom() {
    requiredSize_ = (sq::SizeType)-1;
    internalBufSize_ = (sq::SizeType)-1;
    d_buffer_ = NULL;
    d_randStates_ = NULL;
    d_kernelParams_ = NULL;
    begin_ = end_ = 0;
}

DeviceRandom::~DeviceRandom() {
    if (d_buffer_ != NULL)
        deallocate();
}

void DeviceRandom::assignDevice(Device &device, DeviceStream *devStream) {
    devAlloc_ = device.objectAllocator();
    if (devStream == NULL)
        devStream = device.defaultStream();
    stream_ = devStream->getCudaStream();
}


void DeviceRandom::setRequiredSize(sqaod::SizeType requiredSize) {
    assert(d_buffer_ == NULL);
    requiredSize_ = requiredSize;
    /* Should give 2 chunks, 1 is for roundUp(), other is not to make size == 0 when filled up. */
    internalBufSize_ = roundUp(requiredSize_, (sqaod::SizeType)randGenSize)+ randGenSize * 2;
}
        

void DeviceRandom::deallocate() {
    assert(d_buffer_ != NULL);
    devAlloc_->deallocate(d_buffer_);
    devAlloc_->deallocate(d_randStates_);
    devAlloc_->deallocate(d_kernelParams_);

    d_buffer_ = NULL;
    d_randStates_ = NULL;
    d_kernelParams_ = NULL;
}

void DeviceRandom::seed(unsigned long long seed) {
    if (d_buffer_ != NULL)
        deallocate();
    devAlloc_->allocate(&d_buffer_, internalBufSize_);
    devAlloc_->allocate(&d_randStates_, CURAND_NUM_MTGP32_PARAMS);
    devAlloc_->allocate(&d_kernelParams_, CURAND_NUM_MTGP32_PARAMS);
    /* synchronous */
    throwOnError(hiprandMakeMTGP32KernelState(
                         d_randStates_, MTGPDC_PARAM_TABLE,
                         d_kernelParams_, CURAND_NUM_MTGP32_PARAMS, seed));
}

void DeviceRandom::seed() {
    seed((unsigned long)time(NULL));
}

sqaod::SizeType DeviceRandom::getNRands() const {
    return (end_ - begin_ + internalBufSize_) % internalBufSize_;
}


__global__
static void genRandKernel(int *d_buffer, int offset, int nNums, int bufLen,
                          hiprandStateMtgp32_t *d_state) {
    /* bufLen must be 2^n */
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    offset = (offset + gid) % bufLen;
    for (int idx = 0; idx < nNums; idx += randGenSize) {
        int r = hiprand(&d_state[blockIdx.x]);
        d_buffer[offset] = r;
        offset = (offset + randGenSize) % bufLen;
    }
}


void DeviceRandom::generate() {
    int nToGenerate = requiredSize_ - getNRands();
    nToGenerate = roundUp(nToGenerate, (int)randGenSize);
    if (0 <= nToGenerate) {
        genRandKernel<<<CURAND_NUM_MTGP32_PARAMS, THREAD_NUM, 0, stream_>>>
                (d_buffer_, end_, nToGenerate, internalBufSize_, d_randStates_);
        DEBUG_SYNC;
        end_ = (end_ + nToGenerate) % internalBufSize_;
    }
}

const int *DeviceRandom::get(sqaod::SizeType nRands,
                             sqaod::IdxType *offset, sqaod::SizeType *posToWrap, int alignment) {
    nRands = roundUp(nRands, (sq::SizeType)alignment);
    if (getNRands() < nRands)
        generate();
    assert(nRands <= getNRands());

    *offset = begin_;
    *posToWrap = internalBufSize_;
    begin_ = (begin_ + nRands) % internalBufSize_;
    return d_buffer_;
}

void DeviceRandom::synchronize() {
    throwOnError(hipStreamSynchronize(stream_));
}
