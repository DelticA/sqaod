#include "hip/hip_runtime.h"
#include "DeviceRandomMTGP32.h"
#include "cudafuncs.h"
#include <hiprand/hiprand_mtgp32_host.h>
#include <hiprand/hiprand_kernel.h>
#include <common/Random.h>


/* FIXME: use multiple states utilize more threads.
 *         use memory store */

using namespace sqaod_cuda;
    
enum {
    randGenSize = CURAND_NUM_MTGP32_PARAMS * THREAD_NUM
};


DeviceRandomMTGP32::DeviceRandomMTGP32(Device &device, DeviceStream *devStream) {
    requiredSize_ = -1;
    internalBufSize_ = -1;
    d_buffer_ = NULL;
    d_randStates_ = NULL;
    d_kernelParams_ = NULL;
    begin_ = end_ = 0;
    assignDevice(device, devStream);
}

DeviceRandomMTGP32::DeviceRandomMTGP32() {
    requiredSize_ = -1;
    internalBufSize_ = -1;
    d_buffer_ = NULL;
    d_randStates_ = NULL;
    d_kernelParams_ = NULL;
    begin_ = end_ = 0;
}

DeviceRandomMTGP32::~DeviceRandomMTGP32() {
    if (d_buffer_ != NULL)
        deallocate();
}

void DeviceRandomMTGP32::assignDevice(Device &device, DeviceStream *devStream) {
    devAlloc_ = device.objectAllocator();
    if (devStream == NULL)
        devStream = device.defaultStream();
    stream_ = devStream->getCudaStream();
}


void DeviceRandomMTGP32::setRequiredSize(sq::SizeType requiredSize) {
    /* Should give 2 chunks, 1 is for roundUp(), other is not to make size == 0 when filled up. */
    int newInternalBufSize = roundUp(requiredSize, randGenSize)+ randGenSize * 2;
    if (newInternalBufSize != internalBufSize_) {
        internalBufSize_ = newInternalBufSize;
        if (d_buffer_ != NULL)
            devAlloc_->deallocate(d_buffer_);
        d_buffer_ = NULL;
    }
    requiredSize_ = requiredSize;
}
        

void DeviceRandomMTGP32::deallocate() {
    deallocateStates();
    deallocateBuffer();
}

void DeviceRandomMTGP32::deallocateStates() {
    devAlloc_->deallocate(d_randStates_);
    devAlloc_->deallocate(d_kernelParams_);
    d_randStates_ = NULL;
    d_kernelParams_ = NULL;
}

void DeviceRandomMTGP32::deallocateBuffer() {
    devAlloc_->deallocate(d_buffer_);
    d_buffer_ = NULL;
}

void DeviceRandomMTGP32::seed(unsigned int seed) {
    if (d_randStates_ != NULL)
        deallocateStates();
    devAlloc_->allocate(&d_randStates_, CURAND_NUM_MTGP32_PARAMS);
    devAlloc_->allocate(&d_kernelParams_, CURAND_NUM_MTGP32_PARAMS);
    /* synchronous */
    throwOnError(hiprandMakeMTGP32KernelState(
                         d_randStates_, MTGPDC_PARAM_TABLE,
                         d_kernelParams_, CURAND_NUM_MTGP32_PARAMS, seed));
}

void DeviceRandomMTGP32::seed() {
    seed((unsigned long)time(NULL));
}

sq::SizeType DeviceRandomMTGP32::getNRands() const {
    return (end_ - begin_ + internalBufSize_) % internalBufSize_;
}


__global__
static void genRandKernel(unsigned int *d_buffer, int offset, int nNums, int bufLen,
                          hiprandStateMtgp32_t *d_state) {
    /* bufLen must be 2^n */
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    offset = (offset + gid) % bufLen;
    for (int idx = 0; idx < nNums; idx += randGenSize) {
        unsigned int r = hiprand(&d_state[blockIdx.x]);
        d_buffer[offset] = r;
        offset = (offset + randGenSize) % bufLen;
    }
}


void DeviceRandomMTGP32::generate() {
    throwErrorIf(internalBufSize_ == -1, "DeviceRandom not initialized.");
    if (d_buffer_ == NULL)
        devAlloc_->allocate(&d_buffer_, internalBufSize_);

    int nToGenerate = requiredSize_ - getNRands();
    nToGenerate = roundUp(nToGenerate, (int)randGenSize);
    if (0 <= nToGenerate) {
#if 1
        genRandKernel<<<CURAND_NUM_MTGP32_PARAMS, THREAD_NUM, 0, stream_>>>
                (d_buffer_, end_, nToGenerate, internalBufSize_, d_randStates_);
        DEBUG_SYNC;
#else
        /* generate random numbers on CPU for validation. */
        synchronize();
        for (int idx = 0; idx < nToGenerate; ++idx)
            d_buffer_[(end_ + idx) % internalBufSize_] = sq::random.randInt32();
#endif
        end_ = (end_ + nToGenerate) % internalBufSize_;
    }
}

const unsigned int *DeviceRandomMTGP32::get(sq::SizeType nRands,
                                            sq::IdxType *offset, sq::SizeType *posToWrap, int alignment) {
    nRands = roundUp(nRands, alignment);
    if (getNRands() < nRands)
        generate();
    assert(nRands <= getNRands());

    *offset = begin_;
    *posToWrap = internalBufSize_;
    begin_ = (begin_ + nRands) % internalBufSize_;
    return d_buffer_;
}

void DeviceRandomMTGP32::synchronize() {
    throwOnError(hipStreamSynchronize(stream_));
}
