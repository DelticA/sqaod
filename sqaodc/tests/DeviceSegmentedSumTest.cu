#include "DeviceSegmentedSumTest.h"
#include <hip/hip_runtime.h>
#include <cuda/DeviceSegmentedSum.cuh>
#include <cuda/DeviceCopy.h>
//#include <common/Matrix.h>
#include "utils.h"

namespace sqcu = sqaod_cuda;

DeviceSegmentedSumTest::DeviceSegmentedSumTest(void) : MinimalTestSuite("DeviceSegmentedSumTest")
{
}


DeviceSegmentedSumTest::~DeviceSegmentedSumTest(void)
{
}


void DeviceSegmentedSumTest::setUp() {
    device_.useManagedMemory(true);
    device_.enableLocalStore(false);
    device_.initialize();
}

void DeviceSegmentedSumTest::tearDown() {
    device_.finalize();
}

template<class V>
void DeviceSegmentedSumTest::runSegmentedSum(int segLen, int nSegments) {
    sqcu::DeviceObjectAllocator *alloc = device_.objectAllocator();
    sqcu::DeviceCopy copy(device_);

    typedef sqcu::DeviceMatrixType<V> DeviceMatrix;
    typedef sqcu::DeviceVectorType<V> DeviceVector;
    typedef sq::MatrixType<V> HostMatrix;
    typedef sq::VectorType<V> HostVector;

    typedef sqcu::DeviceSegmentedSumTypeImpl<V, V*, V*, sqcu::Linear> SegmentedSum;

    testcase("SegmentedSum") {
        SegmentedSum segSum(device_);
        segSum.configure(segLen, nSegments, false);
        sq::Dim dim(nSegments, segLen);

        DeviceMatrix dA;
        DeviceVector dx;

        HostMatrix A = testMatBalanced<V>(dim);
        HostVector x = segmentedSum(A, segLen, nSegments);

        copy(&dA, A);
        alloc->allocate(&dx, nSegments);
        segSum.configure(segLen, nSegments, false);
        segSum(dA.d_data, dx.d_data, sqcu::Linear(segLen, 0));
        device_.synchronize();

        TEST_ASSERT(allclose(dx, x, epusiron<V>()));

        alloc->deallocate(dA);
        alloc->deallocate(dx);
    }
}

template<class V>
void DeviceSegmentedSumTest::test() {
#if 1
    typedef sqcu::DeviceSegmentedSumTypeImpl<V, V*, V*, sqcu::Linear> SegmentedSum;
    SegmentedSum segSum(device_);
    for (typename SegmentedSum::MethodMap::iterator it = segSum.methodMap_.begin();
        it != segSum.methodMap_.end(); ++it) {
        runSegmentedSum<V>(it->first, it->first / 8);
    }
#else
    runSegmentedSum<V>(12288, 2);
#endif
}

void DeviceSegmentedSumTest::run(std::ostream &ostm) {
    test<float>();
    test<double>();
}
